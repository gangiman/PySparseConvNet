#include "hip/hip_runtime.h"
#include "SparseConvNetCUDA.h"
#include <iostream>
#include <fstream>
#include <string>
#include <vector>
#include <chrono>
#include <cassert>
#include <algorithm>
#include <sstream>
#include "utilities.h"
#include "SigmoidLayer.h"
#include "NetworkInNetworkLayer.h"
#include "NetworkInNetworkPReLULayer.h"
#include "ConvolutionalLayer.h"
#include "ReallyConvolutionalLayer.h"
#include "ConvolutionalTriangularLayer.h"
#include "MaxPoolingLayer.h"
#include "MaxPoolingTriangularLayer.h"
#include "TerminalPoolingLayer.h"
#include "IndexLearnerLayer.h"
#include "SoftmaxClassifier.h"
#include "BatchProducer.h"
#include "SpatiallySparseDataset.h"
#include <chrono>

SparseConvNetCUDA::SparseConvNetCUDA(int dimension, int nInputFeatures,
                                     int nClasses, int pciBusID, int nTop,
                                     int nBatchProducerThreads)
    : deviceID(initializeGPU(pciBusID)), dimension(dimension),
      nInputFeatures(nInputFeatures), nClasses(nClasses), nTop(nTop),
      nBatchProducerThreads(nBatchProducerThreads) {
  assert(nBatchProducerThreads <= N_MAX_BATCH_PRODUCER_THREADS);
  /* std::cout << "Sparse CNN - dimension=" << dimension
            << " nInputFeatures=" << nInputFeatures << " nClasses=" << nClasses
            << " nThreads=" << nBatchProducerThreads
            << std::endl; */
  nOutputFeatures = nInputFeatures;
  // Set up a pool of SpatiallySparseBatches
  for (int c = 0; c < nBatchProducerThreads; c++) {
    initialSubInterfaces.push_back(new SpatiallySparseBatchSubInterface());
    batchPool.emplace_back(initialSubInterfaces.back());
  }
  cublasError(hipblasCreate(&cublasHandle), __FILE__, __LINE__);
  cublasError(hipblasSetStream(cublasHandle, memStream.stream));
}
SparseConvNetCUDA::~SparseConvNetCUDA() {
  for (auto p : initialSubInterfaces)
    delete p;
  for (auto p : sharedSubInterfaces)
    delete p;
  cublasError(hipblasDestroy(cublasHandle), __FILE__, __LINE__);
}
void SparseConvNetCUDA::addLearntLayer(int nFeatures,
                                       ActivationFunction activationFn,
                                       float dropout, float alpha) {
  if (activationFn != SOFTMAX)
    nFeatures = std::max(KERNELBLOCKSIZE, intRound(nFeatures, KERNELBLOCKSIZE));
  if (dropout > 0)
    dropout = 1 -
              (intRound(nFeatures * (1 - dropout), KERNELBLOCKSIZE) + 0.01f) *
                  1.0f / nFeatures;
  if (dropout < 0)
    dropout = 0;
  /* std::cout << layers.size() << ":"; */
  if (activationFn == PRELU)
    layers.push_back(new NetworkInNetworkPReLULayer(
        memStream, cublasHandle, nOutputFeatures, nFeatures, dropout, alpha));
  else
    layers.push_back(new NetworkInNetworkLayer(memStream, cublasHandle,
                                               nOutputFeatures, nFeatures,
                                               dropout, activationFn, alpha));
  nOutputFeatures = nFeatures;
}
void SparseConvNetCUDA::addNetworkInNetworkLayer(
    int nFeatures, ActivationFunction activationFn, float dropout) {
  addLearntLayer(nFeatures, activationFn, dropout, 1.0f);
}
void SparseConvNetCUDA::addConvolutionalLayer(int nFeatures, int filterSize,
                                              int filterStride,
                                              ActivationFunction activationFn,
                                              float dropout,
                                              int minActiveInputs,
                                              float poolingToFollow) {
  if (false and layers.size() == 0) { // Use for 0-th layer??
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new ReallyConvolutionalLayer(
        memStream, nOutputFeatures, nFeatures, filterSize, filterStride,
        dimension, activationFn, dropout, minActiveInputs, poolingToFollow));
    nOutputFeatures = nFeatures;
  } else {
    if (filterSize > 1) {
      /* std::cout << layers.size() << ":"; */
      layers.push_back(
          new ConvolutionalLayer(memStream, filterSize, filterStride, dimension,
                                 nOutputFeatures, minActiveInputs));
      nOutputFeatures *= ipow(filterSize, dimension);
    }
    addLearntLayer(nFeatures, activationFn, dropout,
                   powf(filterSize * 1.0 / filterStride / poolingToFollow, 2));
  }
}
void SparseConvNetCUDA::addLeNetLayerMP(int nFeatures, int filterSize,
                                        int filterStride, int poolSize,
                                        int poolStride,
                                        ActivationFunction activationFn,
                                        float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, poolSize);
  if (poolSize > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(
        new MaxPoolingLayer(memStream, poolSize, poolStride, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerROFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new RandomOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerPOFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new PseudorandomOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerRDFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new RandomNonOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}
void SparseConvNetCUDA::addLeNetLayerPDFMP(int nFeatures, int filterSize,
                                           int filterStride, int poolSize,
                                           float fmpShrink,
                                           ActivationFunction activationFn,
                                           float dropout, int minActiveInputs) {
  addConvolutionalLayer(nFeatures, filterSize, filterStride, activationFn,
                        dropout, minActiveInputs, fmpShrink);
  if (fmpShrink > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new PseudorandomNonOverlappingFractionalMaxPoolingLayer(
        memStream, poolSize, fmpShrink, dimension));
  }
}

void SparseConvNetCUDA::addTriangularConvolutionalLayer(
    int nFeatures, int filterSize, int filterStride,
    ActivationFunction activationFn, float dropout, int minActiveInputs,
    float poolingToFollow) {
  if (filterSize > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new ConvolutionalTriangularLayer(
        memStream, filterSize, filterStride, dimension, nOutputFeatures,
        minActiveInputs));
    nOutputFeatures *= triangleSize(filterSize, dimension);
  }
  addLearntLayer(nFeatures, activationFn, dropout,
                 powf(filterSize * 1.0 / filterStride / poolingToFollow, 2));
}
void SparseConvNetCUDA::addTriangularLeNetLayerMP(
    int nFeatures, int filterSize, int filterStride, int poolSize,
    int poolStride, ActivationFunction activationFn, float dropout,
    int minActiveInputs) {
  addTriangularConvolutionalLayer(nFeatures, filterSize, filterStride,
                                  activationFn, dropout, poolSize,
                                  minActiveInputs);
  if (poolSize > 1) {
    /* std::cout << layers.size() << ":"; */
    layers.push_back(new MaxPoolingTriangularLayer(memStream, poolSize,
                                                   poolStride, dimension));
  }
}

void SparseConvNetCUDA::addTerminalPoolingLayer(int poolSize, int S) {
  /* std::cout << layers.size() << ":"; */
  layers.push_back(new TerminalPoolingLayer(memStream, poolSize, S));
}

int SparseConvNetCUDA::computeInputSpatialSize(int outputSpatialSize) {
  inputSpatialSize = outputSpatialSize;
  for (int i = layers.size() - 1; i >= 0; i--) {
    inputSpatialSize = layers[i]->calculateInputSpatialSize(inputSpatialSize);
  }
  return inputSpatialSize;
};

void SparseConvNetCUDA::addSoftmaxLayer() {
  addLearntLayer(nClasses, SOFTMAX, 0.0f, 1);
  inputSpatialSize = 1;
  /* std::cout << "Spatially sparse CNN with layer sizes: " << inputSpatialSize; */
  for (int i = layers.size() - 1; i >= 0; i--) {
    inputSpatialSize = layers[i]->calculateInputSpatialSize(inputSpatialSize);
  }
  /* std::cout << std::endl; */
  /* std::cout << "Input-field dimensions = " << inputSpatialSize; */
  // for (int i = 1; i < dimension; ++i)
    /* std::cout << "x" << inputSpatialSize; */
  /* std::cout << std::endl; */
}
void SparseConvNetCUDA::addIndexLearnerLayer() {
  /* std::cout << layers.size() << ":"; */
  layers.push_back(new IndexLearnerLayer(memStream, cublasHandle,
                                         nOutputFeatures, nClasses));
  /* std::cout << "Index Learner " << nOutputFeatures << "-> " << nClasses
            << std::endl; */
  nOutputFeatures = nClasses; // "nClasses"=trainingSet.pictures.size()
  inputSpatialSize = 1;
  for (int i = layers.size() - 1; i >= 0; i--) {
    inputSpatialSize = layers[i]->calculateInputSpatialSize(inputSpatialSize);
  }
  /* std::cout << "Spatially sparse CNN: input size " << inputSpatialSize; */
  // for (int i = 1; i < dimension; ++i)
    /* std::cout << "x" << inputSpatialSize; */
  /* std::cout << std::endl; */
}
void SparseConvNetCUDA::processBatch(SpatiallySparseBatch &batch,
                                     float learningRate, float momentum,
                                     std::ofstream &f, std::ofstream &g) {
  if (batch.type == RESCALEBATCH) {
    float scalingUnderneath = 1;
    for (int i = 0; i < layers.size(); i++) {
      batch.interfaces[i + 1].sub->reset();
      layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
      /* std::cout << i << ":"
                << batch.interfaces[i].sub->features.size() * sizeof(float) /
                       (1 << 20) << "MB "; */
      layers[i]->scaleWeights(batch.interfaces[i], batch.interfaces[i + 1],
                              scalingUnderneath, i == layers.size() - 1);
    }
  } else {
    for (int i = 0; i < layers.size(); i++) {
      batch.interfaces[i + 1].sub->reset();
      layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
    }
  }
  SoftmaxClassifier(batch.interfaces.back(), batch, nTop, memStream);
  if (batch.type == TRAINBATCH)
    for (int i = layers.size() - 1; i >= 0; i--) {
      layers[i]->backwards(batch, batch.interfaces[i], batch.interfaces[i + 1],
                           learningRate, momentum);
    }
  if (f)
    for (int j = 0; j < batch.predictions.size(); j++) {
      for (int k = 0; k < batch.predictions[j].size(); k++) {
        if (k > 0)
          f << " ";
        f << batch.predictions[j][k];
      }
      f << std::endl;
    }
  if (g)
    for (int j = 0; j < batch.predictions.size(); j++) {
      for (int k = 0; k < batch.probabilities[j].size(); k++) {
        if (k > 0)
          g << " ";
        g << batch.probabilities[j][k];
      }
      g << std::endl;
    }
}

activation SparseConvNetCUDA::processBatchForward(SpatiallySparseBatch &batch) {
    for (int i = 0; i < layers.size(); i++) {
      batch.interfaces[i + 1].sub->reset();
      layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
    }

    SpatiallySparseBatchInterface &last_layer_batch_inteface = batch.interfaces.back();
    last_layer_batch_inteface.sub->features.copyToCPUAsync(memStream);
    const std::vector<float> features = last_layer_batch_inteface.sub->features.hVector();
    //std::copy(features.cbegin(), features.cend(), std::ostream_iterator(std::cout, ","));
      activation last_layer_activation;
      last_layer_activation.grid_size = last_layer_batch_inteface.grids[0].mp.size();
      last_layer_activation.feature_size = features.size();
      last_layer_activation.nSpatialSites = last_layer_batch_inteface.nSpatialSites;
      last_layer_activation.spatialSize = last_layer_batch_inteface.spatialSize;
      last_layer_activation.nFeatures = last_layer_batch_inteface.nFeatures;
      last_layer_activation.features = features;

      for (SparseGridMap::iterator it = last_layer_batch_inteface.grids[0].mp.begin();
          it != last_layer_batch_inteface.grids[0].mp.end(); ++it) {
        last_layer_activation.sparse_grid.push_back(std::make_pair(it->first, it->second));
      }

    last_layer_batch_inteface.sub->features.copyToGPUAsync(memStream);
    return last_layer_activation;
}

void SparseConvNetCUDA::processBatchBackward(SpatiallySparseBatch &batch,
                                     float learningRate, float momentum,
                                     const std::vector<float> &dfeatures) {

  SpatiallySparseBatchInterface &input = batch.interfaces.back();

  assert(batch.type == TRAINBATCH);
  assert(batch.batchSize == input.nSpatialSites);
  assert(input.nFeatures == input.featuresPresent.size());

  input.sub->dfeatures.copyToCPU();
  input.sub->dfeatures.resize(input.nSpatialSites *
                                input.featuresPresent.size());

  input.sub->dfeatures.hVector() = dfeatures;
  input.sub->dfeatures.copyToGPU();

  cudaCheckError();

  for (int i = layers.size() - 1; i >= 0; i--) {
    layers[i]->backwards(batch, batch.interfaces[i], batch.interfaces[i + 1],
                         learningRate, momentum);
  }
}

pd_report SparseConvNetCUDA::processDataset(SpatiallySparseDataset &dataset,
                                        int batchSize, float learningRate,
                                        float momentum) {
  assert(dataset.pictures.size() > 0);
  float errorRate = 0, nll = 0;
  multiplyAddCount = 0;
  auto start = std::chrono::system_clock::now();
  std::ofstream f, g;
  BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
  if (dataset.type == UNLABELEDBATCH) {
    f.open("unlabelledData.predictions");
    g.open("unlabelledData.probabilities");
  }
  int batch_counter = 0;
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    // std::cout << "batch: " << batch_counter << std::endl;
    batch_counter++;
    processBatch(*batch, learningRate, momentum, f, g);
    errorRate += batch->mistakes * 1.0 / dataset.pictures.size();
    nll += batch->negativeLogLikelihood * 1.0 / dataset.pictures.size();
  }
  auto end = std::chrono::system_clock::now();
  auto diff =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

  pd_report report;
  report.errorRate = errorRate;
  report.nll = nll;
  report.MegaMultiplyAdds_per_sample = roundf(multiplyAddCount / dataset.pictures.size() / 1000000);
  report.time = diff / 1000000000L;
  report.GigaMultiplyAdds_per_s = roundf(multiplyAddCount / diff);
  report.rate = roundf(dataset.pictures.size() * 1000000000.0f / diff);
  return report;
}

std::vector<std::vector<float>> SparseConvNetCUDA::predict(
        SpatiallySparseDataset &dataset) {
  assert(dataset.pictures.size() > 0);
  std::vector<std::vector<float>> result_matrix;
//  result_matrix.resize(dataset.pictures.size());
  int batchSize = 1;
  float learningRate, momentum;
  std::ofstream f, g;
  BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    processBatch(*batch, learningRate, momentum, f, g);
    for (int i = 0; i < batch->batchSize; i++) {
      result_matrix.push_back(batch->probabilities[i]);
    }
  }
  return result_matrix;
}

std::vector<struct activation> SparseConvNetCUDA::layer_activations(
        SpatiallySparseDataset &dataset) {
  assert(dataset.pictures.size() == 1);
  int batchSize = 1;
  std::vector<struct activation> activations;
  BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    processBatchForward(*batch);
    for (int i = 0; i < batch->interfaces.size(); ++i){
      batch->interfaces[i].sub->features.copyToCPUAsync(memStream);
      std::vector<float> &features = batch->interfaces[i].sub->features.hVector();
      activations.push_back(activation());
      activations[i].grid_size = batch->interfaces[i].grids[0].mp.size();
      activations[i].feature_size = features.size();
      activations[i].nSpatialSites = batch->interfaces[i].nSpatialSites;
      activations[i].spatialSize = batch->interfaces[i].spatialSize;
      activations[i].nFeatures = batch->interfaces[i].nFeatures;
      activations[i].features = features;
      for (SparseGridMap::iterator it = batch->interfaces[i].grids[0].mp.begin();
          it != batch->interfaces[i].grids[0].mp.end(); ++it) {
        activations[i].sparse_grid.push_back(std::make_pair(it->first, it->second));
      }
    }
  }
  return activations;
}

std::vector<struct pd_report> SparseConvNetCUDA::processDatasetRepeatTest(
    SpatiallySparseDataset &dataset, int batchSize, int nReps,
    std::string predictionsFilename, std::string confusionMatrixFilename) {
  assert(dataset.pictures.size() > 0);
  multiplyAddCount = 0;
  auto start = std::chrono::system_clock::now();
  std::vector<std::vector<int>> votes(dataset.pictures.size());
  std::vector<std::vector<float>> probs(dataset.pictures.size());
  for (int i = 0; i < dataset.pictures.size(); ++i) {
    votes[i].resize(dataset.nClasses);
    probs[i].resize(dataset.nClasses);
  }
  std::vector<struct pd_report> reports;
  for (int rep = 1; rep <= nReps; ++rep) {
    BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
    int batch_counter = 0;
    while (SpatiallySparseBatch *batch = bp.nextBatch()) {
      // std::cout << "batch: " << batch_counter << std::endl;
      std::ofstream f, g;
      processBatch(*batch, 0, 0, f, g);
      for (int i = 0; i < batch->batchSize; ++i) {
        int ii = batch->sampleNumbers[i];
        votes[ii][batch->predictions[i][0]]++;
        for (int j = 0; j < dataset.nClasses; ++j)
          probs[ii][j] += batch->probabilities[i][j];
      }
    }
    int errors = dataset.pictures.size();
    float nll = 0;
    for (int i = 0; i < dataset.pictures.size(); ++i) {
      std::vector<int> predictions = vectorTopIndices(probs[i], nTop);
      for (int j = 0; j < nTop; j++)
        if (predictions[j] == dataset.pictures[i]->label)
          errors--;
      nll -= log(
          std::max(probs[i][dataset.pictures[i]->label] / rep, (float)1.0e-15));
    }

    if (!predictionsFilename.empty()) {
      // std::cout << predictionsFilename << std::endl;
      std::ofstream f(predictionsFilename.c_str());
      if (!dataset.header.empty())
        f << dataset.header << std::endl;
      for (int i = 0; i < dataset.pictures.size(); ++i) {
        f << dataset.pictures[i]->identify();
        if (dataset.type != UNLABELEDBATCH)
          f << "," << dataset.pictures[i]->label;
        for (int j = 0; j < dataset.nClasses; ++j)
          f << "," << probs[i][j] / rep;
        f << std::endl;
      }
    }
    if (!confusionMatrixFilename.empty()) {
      std::vector<float> cm(dataset.nClasses * dataset.nClasses);
      for (int i = 0; i < dataset.pictures.size(); ++i)
        for (int j = 0; j < dataset.nClasses; ++j)
          cm[dataset.pictures[i]->label * dataset.nClasses + j] +=
              probs[i][j] / rep;
      std::ofstream f(confusionMatrixFilename.c_str());
      for (int i = 0; i < dataset.nClasses; ++i) {
        for (int j = 0; j < dataset.nClasses; ++j) {
          f << cm[i * dataset.nClasses + j] << " ";
        }
        f << std::endl;
      }
    }
    auto end = std::chrono::system_clock::now();
    auto diff = std::chrono::duration_cast<std::chrono::nanoseconds>(
                    end - start).count();
    // buffer.clear();
    reports.push_back(pd_report());
    reports[rep - 1].errorRate = 1.0 * errors / dataset.pictures.size();
    reports[rep - 1].nll = nll / dataset.pictures.size();
    reports[rep - 1].MegaMultiplyAdds_per_sample = roundf(multiplyAddCount / dataset.pictures.size() / 1000000);
    reports[rep - 1].time = diff / 1000000000L;
    reports[rep - 1].GigaMultiplyAdds_per_s = roundf(multiplyAddCount / diff);
    reports[rep - 1].rate = roundf(dataset.pictures.size() * 1000000000.0f / diff);
  }
  return reports;
}
void SparseConvNetCUDA::loadWeights(std::string baseName, int epoch,
                                    bool momentum, int firstNlayers) {
  std::string filename = std::string(baseName) + std::string("_epoch-") +
                         std::to_string(epoch) + std::string(".cnn");
  std::ifstream f;
  f.open(filename.c_str(), std::ios::out | std::ios::binary);
  if (f) {
    /* std::cout << "Loading network parameters from " << filename << std::endl; */
  } else {
    std::cout << "Cannot find " << filename << std::endl;
    exit(EXIT_FAILURE);
  }
  for (int i = 0; i < std::min((int)layers.size(), firstNlayers); i++)
    layers[i]->loadWeightsFromStream(f, momentum);
  if (inputNormalizingConstants.size() > 0)
    f.read((char *)&inputNormalizingConstants[0],
           sizeof(float) * inputNormalizingConstants.size());
  f.close();
}
void SparseConvNetCUDA::saveWeights(std::string baseName, int epoch,
                                    bool momentum) {
  std::string filename = std::string(baseName) + std::string("_epoch-") +
                         std::to_string(epoch) + std::string(".cnn");
  std::ofstream f;
  f.open(filename.c_str(), std::ios::binary);
  if (f) {
    for (int i = 0; i < layers.size(); i++)
      layers[i]->putWeightsToStream(f, momentum);
    if (inputNormalizingConstants.size() > 0)
      f.write((char *)&inputNormalizingConstants[0],
              sizeof(float) * inputNormalizingConstants.size());
    f.close();
  } else {
    // std::cout << "Cannot write " << filename << std::endl;
    exit(EXIT_FAILURE);
  }
}
void SparseConvNetCUDA::processIndexLearnerBatch(SpatiallySparseBatch &batch,
                                                 float learningRate,
                                                 float momentum,
                                                 std::ofstream &f) {
  int n = layers.size();
  for (int i = 0; i < n - 1; i++) // Stop 1 early (unless it is a training
                                  // batch)
    layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
  if (f.is_open()) {
    assert(batch.interfaces[n - 1].nFeatures ==
           batch.interfaces[n - 1].featuresPresent.size());
    for (int i = 0; i < batch.batchSize; i++) {
      f << batch.sampleNumbers[i] << " " << batch.labels.hVector()[i];
      for (int j = 0; j < batch.interfaces[n - 1].nFeatures; j++)
        f << " "
          << batch.interfaces[n - 1]
                 .sub->features
                 .hVector()[i * batch.interfaces[n - 1].nFeatures + j];
      f << std::endl;
    }
  }
  if (batch.type == TRAINBATCH) {
    static_cast<IndexLearnerLayer *>(layers[n - 1])->indexLearnerIndices =
        batch.sampleNumbers;
    layers[n - 1]->forwards(batch, batch.interfaces[n - 1],
                            batch.interfaces[n]);
    IndexLearner(batch.interfaces[n], batch, nTop, memStream);
    for (int i = n - 1; i >= 0; i--)
      layers[i]->backwards(batch, batch.interfaces[i], batch.interfaces[i + 1],
                           learningRate, momentum);
  }
}
float SparseConvNetCUDA::processIndexLearnerDataset(
    SpatiallySparseDataset &dataset, int batchSize, float learningRate,
    float momentum) {
  assert(dataset.pictures.size() > 0);
  float errorRate = 0, nll = 0;
  auto start = std::chrono::system_clock::now();
  multiplyAddCount = 0;
  std::ofstream f;
  BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
  if (dataset.type != TRAINBATCH) {
    std::string filename = dataset.name + ".features";
    f.open(filename.c_str());
  }
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    processIndexLearnerBatch(*batch, learningRate, momentum, f);
    errorRate += batch->mistakes * 1.0 / dataset.pictures.size();
    nll += batch->negativeLogLikelihood * 1.0 / dataset.pictures.size();
  }
  auto end = std::chrono::system_clock::now();
  auto diff =
      std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();
  // if (dataset.type == TRAINBATCH)
    /* std::cout << dataset.name << " Mistakes:" << 100 * errorRate
              << "% NLL:" << nll << " MegaMultiplyAdds/sample:"
              << roundf(multiplyAddCount / dataset.pictures.size() / 1000000)
              << " time:" << diff / 1000000000L
              << "s GigaMultiplyAdds/s:" << roundf(multiplyAddCount / diff)
              << " rate:"
              << roundf(dataset.pictures.size() * 1000000000.0f / diff) << "/s"
              << std::endl; */
  return nll;
}
void SparseConvNetCUDA::processBatchDumpTopLevelFeaturess(
    SpatiallySparseBatch &batch, std::ofstream &f) { // editted: test
  int n = layers.size();
  for (int i = 0; i < layers.size() - 1; i++) {
    layers[i]->forwards(batch, batch.interfaces[i], batch.interfaces[i + 1]);
  }
  assert(batch.interfaces[n - 1].nFeatures ==
         batch.interfaces[n - 1].featuresPresent.size());
  for (int i = 0; i < batch.batchSize; i++) {
    f << batch.sampleNumbers[i] << " " << batch.labels.hVector()[i];
    for (int j = 0; j < batch.interfaces[n - 1].nFeatures; j++)
      f << " "
        << batch.interfaces[n - 1]
               .sub->features
               .hVector()[i * batch.interfaces[n - 1].nFeatures + j];
    f << std::endl;
  }
}
void SparseConvNetCUDA::processDatasetDumpTopLevelFeatures(
    SpatiallySparseDataset &dataset, int batchSize, int reps) {
  assert(dataset.pictures.size() > 0);
  std::ofstream f;
  assert(dataset.type != TRAINBATCH);
  std::string filename = dataset.name + ".features";
  f.open(filename.c_str());
  for (int i = 0; i < reps; i++) {
    BatchProducer bp(*this, dataset, inputSpatialSize, batchSize);
    while (SpatiallySparseBatch *batch = bp.nextBatch()) {
      processBatchDumpTopLevelFeaturess(*batch, f);
    }
  }
}

void SparseConvNetCUDA::calculateInputRegularizingConstants(
    SpatiallySparseDataset dataset) { // make copy of the dataset
  inputNormalizingConstants.resize(
      0); // Make sure input features rescaling is turned off.
  /* std::cout << "Using " << std::min(10000, (int)dataset.pictures.size())
            << " out of " << dataset.pictures.size()
            << " training samples to calculate regularizing constants."
            << std::endl; */
  if (dataset.pictures.size() > 10000)
    dataset.pictures.resize(10000);
  dataset.type = TESTBATCH; // pretend it is a test batch to turn off dropout
                            // and training data augmentation
  BatchProducer bp(*this, dataset, inputSpatialSize, 100);
  std::vector<float> c(nInputFeatures, 0);
  while (SpatiallySparseBatch *batch = bp.nextBatch()) {
    batch->interfaces[0].sub->features.copyToCPUAsync(memStream);
    std::vector<float> &features = batch->interfaces[0].sub->features.hVector();
    for (int i = 0; i < features.size(); ++i)
      c[i % nInputFeatures] =
          std::max(c[i % nInputFeatures], std::fabs(features[i]));
  }
  for (int i = 0; i < nInputFeatures; ++i) {
    inputNormalizingConstants.push_back(c[i] > 0 ? 1.0f / c[i] : 0);
    /* std::cout << inputNormalizingConstants.back() << " "; */
  }
  /* std::cout << std::endl; */
}
